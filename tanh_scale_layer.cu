#include "hip/hip_runtime.h"
#include <vector>
#include "caffe/blob.hpp"
#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/tanh_scale_layer.hpp"

namespace caffe {
template <typename Dtype>
__global__ void ScaleFowardGpu(int nthreads,
          const Dtype* bottom_data, Dtype* top_data,
          Dtype* tanhx_) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    tanhx_[index] = tanh(bottom_data[index]);
    top_data[index] = (2 + tanhx_[index]) * bottom_data[index];
  }
}



template <typename Dtype>
void TanhScaleLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  
  const Dtype* bottom_data = bottom[0]->gpu_data();  
  Dtype* top_data = top[0]->mutable_gpu_data();  
  Dtype* tanhx_ = tanh_x.mutable_gpu_data();

  /************************* normalize *************************/

  int nthreads = bottom[0]->count();
  ScaleFowardGpu<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads,                               
                                bottom_data, top_data,
                                tanhx_);
  
}

template <typename Dtype>
__global__ void ScaleBackwardGPU(int nthreads,
const Dtype* top_diff, 
const Dtype* bottom_data, Dtype* bottom_diff,
const Dtype* tanhx_){

  CUDA_KERNEL_LOOP(index, nthreads) {
    bottom_diff[index] = top_diff[index] * (2 + tanhx_[index] + bottom_data[index] * (1- powf(tanhx_[index], 2)));
  }
}

template <typename Dtype>
void TanhScaleLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  
  const Dtype* top_diff = top[0]->cpu_diff();    
  const Dtype* bottom_data = bottom[0]->cpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_cpu_diff();
  Dtype* tanhx_ = tanh_x.mutable_gpu_data();


	if (propagate_down[0]) {    
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();    
		int nthreads = bottom[0]->count();
    ScaleBackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
    CAFFE_CUDA_NUM_THREADS>>>(nthreads, 
    top_diff,
    bottom_data, bottom_diff,
    tanhx_);

	}
  
}

INSTANTIATE_LAYER_GPU_FUNCS(TanhScaleLayer);

}